#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"

#define VALUEBITS 5

bool isEmpty(int cell){
  // returns true if value is not set yet, false otherwise
  int allOnes = (1 << (VALUEBITS+1)) -1;
  return !((cell & allOnes));
}

void eliminateChoices(int *board, int boardSize, int row, int col, int n) {
	//number to eliminate as option from relevant cells
	int num = board[row * boardSize + col] % (1<<VALUEBITS);
	//filter for removing options from cells
	int filter = ~(1 << (VALUEBITS + num));
	for (int rowI = 0; rowI < boardSize; rowI++) {
		//eliminate choices for the column
		if (rowI != row) {
			int i = rowI * boardSize + col;
			board[i] = board[i] & filter; 
		}
	}

	for (int colI = 0; colI < boardSize; colI++) {
		//eliminate choices for the row
		if (colI != col) {
			int i = row * boardSize + colI;
			board[i] = board[i] & filter; 
		}
	}

	/*base row and col for the square the cell is located in
	 (the index of upper-right corner of the square) */
	int baseRow = row / n * n;
	int baseCol = col / n * n;
	for (int squareI = 0; squareI < boardSize; squareI++){
		//eliminate choices for the square
		int squareRow = baseRow + squareI / n;
		int squareCol = baseCol + squareI % n;
		if (squareCol != col || squareRow != row) {
			int i = squareRow * boardSize + squareCol;
			board[i] = board[i] & filter;
		}
	}
}

void initialChoiceElm(int *board, int boardSize, int n) {
	//n is square root of board size
	for (int row = 0; row < boardSize; row++) {
		for (int col = 0; col < boardSize; col++) {
			int i = row * boardSize + col;
			if (board[i] % (1<<VALUEBITS)) {
				//if the cell in the board has a value
				eliminateChoices(board, boardSize, row, col, n);
			}
		}
	}
}

int log2(int n) {
	//REQUIRES: n is a power of 2, n != 0
	int log = 0;
	while (n >>= 1) ++log;
	return log;
}

bool elimination(int *board, int boardSize, bool &cellChanged, int n) {
	//return false iff the board given has no valid solution
	for (int i = 0; i < boardSize * boardSize; i++) {
		int value = board[i];
		if (!(value % (1<<VALUEBITS))) {
			//cell is currently empty
			value = value >> VALUEBITS;
			//all the choice bits are 0, so no solution to board
			if (!value) return false;

			if (!(value & (value - 1))) {
				//value is a power of 2, aka there is only one value this cell can take
				cellChanged = true;
				board[i] += log2(value);
				eliminateChoices(board, boardSize, i / boardSize, i % boardSize, n);
			}
		}
	}
	return true;
}


void loneRanger(int *board, int boardSize, bool &cellChanged, int n) {
	for (int i = 0; i < boardSize * boardSize; i++) {
		int value = board[i];
		if (!(value % (1<<VALUEBITS))) {

			//cell is currently empty
			int row = i / boardSize;
			int col = i % boardSize;

			//create a mask to find which choices the other cells in col have
			int mask = 0;
			for (int rowI = 0; rowI < boardSize; rowI++) {
				if (rowI != row) mask = mask | board[rowI * boardSize + col];
			}

			//mask out the choices of the other cells in col
			value = value & (~mask);

			//if value still has 1 choice
			if (value && !(value & (value - 1))) {
				cellChanged = true;
				//write choice to cell and eliminate choices from relevant cells
				board[i] = value + log2(value) - VALUEBITS;
				eliminateChoices(board, boardSize, row, col, n);
				continue;
			}

			//do the same for rows and blocks
			value = board[i];
			mask = 0;
			for (int colI = 0; colI < boardSize; colI++) {
				if (colI != col) {
					mask = mask | board[row * boardSize + colI];
				}
			}

			value = value & (~mask);

			if (value && !(value & (value - 1))) {
				cellChanged = true;
				board[i] = value + log2(value) - VALUEBITS;
				eliminateChoices(board, boardSize, row, col, n);
				continue;
			}

			/*base row and col for the square the cell is located in
			 (the index of upper-right corner of the square) */
			int baseRow = row / n * n;
			int baseCol = col / n * n;

			value = board[i];
			mask = 0;
			for (int squareI = 0; squareI < boardSize; squareI++){
				int squareRow = baseRow + squareI / n;
				int squareCol = baseCol + squareI % n;
				if (squareCol != col || squareRow != row) {
					mask = mask | board[squareRow * boardSize + squareCol];
				}
			}
			
			value = value & (~mask);

			if (value && !(value & (value - 1))) {
				cellChanged = true;
				board[i] = value + log2(value) - VALUEBITS;
				eliminateChoices(board, boardSize, row, col, n);
			}
		}
	}
}

int getCellOptions(int cellA){
	//returns the options part only 
	return cellA >> VALUEBITS;
}

void setOptions(int *board, int index, int options){
	board[index] = options << VALUEBITS;
}

void removeOption(int *board, int index, int option){
	int mask = ~(1 << (option + VALUEBITS));
	board[index] = board[index] & mask;
}

int hasOption(int cell, int option){
	return cell & (1 << (option + VALUEBITS));
}

void eliminateFromBoxRow(int *board, int boxSize ,int index, int option, bool &choicesChanged){
	for (int col = 0; col < boxSize; col++)
	{
		if (hasOption(board[index + col],option))
		{ 
			choicesChanged = true;
			removeOption(board, index + col, option);

		}
	}
}
void eliminateFromBoxCol(int *board, int boardsize, int boxSize ,int index, int option, bool &choicesChanged){
	for (int row = 0; row < boxSize; row++)
	{
		if (hasOption(board[index + row*boardsize],option))
		{ 
			choicesChanged = true;
			removeOption(board, index + row*boardsize, option);
		}
	}
}

void boxElimination(int *board, int boardSize, bool &choicesChanged, int n){
	for (int bx = 0; bx < n; bx++)
	{
		for (int by = 0; by < n; by++)
		{
			int box_index = bx*n*boardSize + by*n;
			// Compare every 2 distenct cells
			for (int option = 1; option <= boardSize; option++){
				int row = -1; //uninitialized
				int col = -1; //uninitialized
				bool optionOnRow = true;
				bool optionOnCol = true;
				for (int i = 0; i < boardSize; i++){
					int A = ((i/n))*boardSize + (i%n) + box_index;
					int cellA = board[A];
					if (isEmpty(cellA)){ //avoid already set cells
						if (hasOption(cellA, option)){
							if (row == -1) {
								row = i/n;
								col = i%n;
							} 
							if (row != i/n) optionOnRow = false;
							if (col != i%n) optionOnCol = false;
						}
					}
				}
				if (row != -1 && optionOnRow){
					for (int j = 0; j < n; j++)
					{
						if (j != by) eliminateFromBoxRow(board, n, bx*n*boardSize + row*boardSize + j*n,option, choicesChanged);
					}
				}
				if (col != -1 && optionOnCol){
					for (int j = 0; j < n; j++)
					{
						if (j != bx) eliminateFromBoxCol(board, boardSize, n, j*n*boardSize + by*n + col,option, choicesChanged);
					}
				}
			}
		}
	}
}

/*
 * bitCount - returns count of number of 1's in word
 *   Examples: bitCount(5) = 2, bitCount(7) = 3
 * Fast bitcount oshafie 15213's datalab
 */
int bitCount(int x) {
	int everyOtherTemp = (0x55 << 8) +(0x55);
	int everyOther = (everyOtherTemp << 16) +(everyOtherTemp);//6
	int every2Temp = (0x33 << 8) +(0x33);
	int every2 = (every2Temp << 16) + every2Temp;
	int every4Temp = (0xf << 8) +(0xf);
	int every4 = (every4Temp << 16) + every4Temp;//6
	int every8 = (0xff << 16) +(0xff);//2
	int every16 = (0xff << 8) +(0xff);//2
	x = (x & everyOther) + ((x >> 1) & everyOther);//4
	x = (x & every2) + ((x >> 2) & every2);//4
	x = (x & every4) + ((x >> 4) & every4);//4
	x = (x & every8) + ((x >> 8) & every8);//4
	return (x & every16) + ((x >> 16)& every16);//4
}

void twinsInRow(int *board, int boardSize, bool &choicesChanged){
	// We could assume that every empty cell has at least 2 options, therefore no need for filtering (2+)-option cells
	int A;
	int cellA;
	int B;
	int cellB;
	// Find twins per row
	for (int r = 0; r < boardSize; r++)
	{
		// Compare every 2 distenct cells
		for (int i = 0; i < boardSize-1; i++)
		{
			A = (r*boardSize) +i;
			cellA = board[A];
			if (isEmpty(cellA)){ //avoid already set cells
				cellA = getCellOptions(cellA);
				for (int j = i+1; j < boardSize; j++)
				{
					B = (r*boardSize)+j;
					cellB = board[B];
					if (isEmpty(cellB)){ //avoid already set cells
						cellB = getCellOptions(cellB);
						// cellA and cellB are twins if cellA & cellB has exactly 2 options
						int options = cellA & cellB;
						if (bitCount(options) >= 2){
							//Check that no other cell have the any of the 2 options available
							int optionsUnion = 0; // get all options for all left cells
							for (int t = 0; t < boardSize; t++)
							{
								if (t != i && t != j){
									int tempCell = board[(r*boardSize)+t];
									optionsUnion = optionsUnion | getCellOptions(tempCell);
								}
							}
							options = options & ~optionsUnion; // remove options that are in the union
							if (bitCount(options) == 2){
								if (options != cellA){
									setOptions(board, A, options);
									choicesChanged = true;
								}
								if (options != cellB){
									setOptions(board, B, options);
									choicesChanged = true;
								}
							}
						}
					}
				}
			}
		}
	}
}

void twinsInBox(int *board, int boardSize, bool &choicesChanged){
	// We could assume that every empty cell has at least 2 options, therefore no need for filtering (2+)-option cells
	int A;
	int cellA;
	int B;
	int cellB;
	int n = sqrt(boardSize);
	// Find twins per box
	for (int bx = 0; bx < n; bx++)
	{
		for (int by = 0; by < n; by++)
		{
			int box_index = bx*n*boardSize + by*n;
			// Compare every 2 distenct cells
			for (int i = 0; i < boardSize -1; i++)
			{
				A = ((i/n))*boardSize + (i%n) + box_index;
				cellA = board[A];
				if (isEmpty(cellA)){ //avoid already set cells
					cellA = getCellOptions(cellA);
					for (int j = i+1; j < boardSize; j++)
					{
						B = ((j/n))*boardSize + (j%n) + box_index;
						cellB = board[B];
						if (isEmpty(cellB)){ //avoid already set cells
							cellB = getCellOptions(cellB);
							// cellA and cellB are twins if cellA & cellB has exactly 2 options
							int options = cellA & cellB;
							if (bitCount(options) >= 2){
								//Check that no other cell have the any of the 2 options available
								int optionsUnion = 0; // get all options for all left cells
								for (int t = 0; t < boardSize; t++)
								{
									if (t != i && t != j){
										int tempCell = board[((t/n))*boardSize + (t%n) + box_index];
										optionsUnion = optionsUnion | getCellOptions(tempCell);
									}
								}
								options = options & ~optionsUnion; // remove options that are in the union
								if (bitCount(options) == 2){
									if (options != cellA){
									setOptions(board, A, options);
									choicesChanged = true;
									}
									if (options != cellB){
										setOptions(board, B, options);
										choicesChanged = true;
									}
								}
							}
						}
					}
				}
			}
		}
	}
}

void twinsInColumn(int *board, int boardSize, bool &choicesChanged){
	// We could assume that every empty cell has at least 2 options, therefore no need for filtering (2+)-option cells
	int A;
	int cellA;
	int B;
	int cellB;
	// Find twins per column
	for (int c = 0; c < boardSize; c++)
	{
		// Compare every 2 distenct cells
		for (int i = 0; i < boardSize-1; i++)
		{
			A = i*boardSize + c;
			cellA = board[A];
			if (isEmpty(cellA)){ //avoid already set cells
				cellA = getCellOptions(cellA);
				for (int j = i+1; j < boardSize; j++)
				{
					B = j*boardSize + c;
					cellB = board[B];
					if (isEmpty(cellB)){ //avoid already set cells
						cellB = getCellOptions(cellB);
						// cellA and cellB are twins if cellA & cellB has exactly 2 options
						int options = cellA & cellB;
						if (bitCount(options) >= 2){
							//Check that no other cell have the any of the 2 options available
							int optionsUnion = 0; // get all options for all left cells
							for (int t = 0; t < boardSize; t++)
							{
								if (t != i && t != j){
									int tempCell = board[t*boardSize + c];
									optionsUnion = optionsUnion | getCellOptions(tempCell);
								}
							}
							options = options & ~optionsUnion; // remove options that are in the union
							if (bitCount(options) == 2){
								if (options != cellA){
									setOptions(board, A, options);
									choicesChanged = true;
								}
								if (options != cellB){
									setOptions(board, B, options);
									choicesChanged = true;
								}
							}
						}
					}
				}
			}
		}
	}
}

void tripletsInRow(int *board, int boardSize, bool &choicesChanged){
	// We could assume that every empty cell has at least 2 options, therefore no need for filtering (2+)-option cells
	int A;
	int cellA;
	int B;
	int cellB;
	int C;
	int cellC;
	// Find twins per row
	for (int r = 0; r < boardSize; r ++)
	{
		// Compare every 2 distenct cells
		for (int i = 0; i < boardSize-2; i++)
		{
			A = r*boardSize+i;
			cellA = board[A];
			if (isEmpty(cellA)){ //avoid already set cells
				cellA = getCellOptions(cellA);
				for (int j = i+1; j < boardSize-1; j++)
				{
					B = r*boardSize+j;
					cellB = board[B];
					if (isEmpty(cellB)){ //avoid already set cells
						cellB = getCellOptions(cellB);
						for (int k = j+1; k < boardSize; k++)
						{
							C = r*boardSize+k;
							cellC = board[C];
							if (isEmpty(cellC)){ //avoid already set cells
								cellC = getCellOptions(cellC);
								// cellA, cellB and cellC are triplets if (cellA & cellB & cellC) has exactly 3 options
								int options = cellA & cellB & cellC;
								if (bitCount(options) >= 3){
									int optionsUnion = 0; // get all options for all left cells
									for (int t = 0; t < boardSize; t++)
									{
										if (t != i && t != j){
											int tempCell = board[r*boardSize+t];
											optionsUnion = optionsUnion | getCellOptions(tempCell);
										}
									}
									options = options & ~optionsUnion; // remove options that are in the union
									if (bitCount(options) == 3){
										if (options != cellA){
											setOptions(board, A, options);
											choicesChanged = true;
										}
										if (options != cellB){
											setOptions(board, B, options);
											choicesChanged = true;
										}
										if (options != cellC){
											setOptions(board, C, options);
											choicesChanged = true;
										}
									}
								}
							}
						}
					}
				}
			}
		}
	}
}

void tripletsInBox(int *board, int boardSize, bool &choicesChanged){
	// We could assume that every empty cell has at least 2 options, therefore no need for filtering (2+)-option cells
	int A;
	int cellA;
	int B;
	int cellB;
	int C;
	int cellC;
	int n = sqrt(boardSize);
	// Find twins per box
	for (int bx = 0; bx < n; bx++)
	{
		for (int by = 0; by < n; by++)
		{
			int box_index = bx*n*boardSize + by*n;
			// Compare every 2 distenct cells
			for (int i = 0; i < boardSize -2; i++)
			{
				A = ((i/n))*boardSize + (i%n) + box_index;
				cellA = board[A];
				if (isEmpty(cellA)){ //avoid already set cells
					cellA = getCellOptions(cellA);
					for (int j = i+1; j < boardSize -1; j++)
					{
						B = ((j/n))*boardSize + (j%n) + box_index;
						cellB = board[B];
						if (isEmpty(cellB)){ //avoid already set cells
							cellB = getCellOptions(cellB);
							for (int k = j+1; k < boardSize; k++)
							{
								C = ((k/n))*boardSize + (k%n) + box_index;
								cellC = board[C];
								if (isEmpty(cellC)){ //avoid already set cells
									cellC = getCellOptions(cellC);
									// cellA, cellB and cellC are triplets if (cellA & cellB & cellC) has exactly 3 options
									int options = cellA & cellB & cellC;
									if (bitCount(options) >= 3){
										int optionsUnion = 0; // get all options for all left cells
										for (int t = 0; t < boardSize; t++)
										{
											if (t != i && t != j){
												int tempCell = board[((t/n))*boardSize + (t%n) + box_index];
												optionsUnion = optionsUnion | getCellOptions(tempCell);
											}
										}
										options = options & ~optionsUnion; // remove options that are in the union
										if (bitCount(options) == 3){
											if (options != cellA){
												setOptions(board, A, options);
												choicesChanged = true;
											}
											if (options != cellB){
												setOptions(board, B, options);
												choicesChanged = true;
											}
											if (options != cellC){
												setOptions(board, C, options);
												choicesChanged = true;
											}
										}
									}
								}
							}
						}
					}
				}
			}
		}
	}
}
void tripletsInColumn(int *board, int boardSize, bool &choicesChanged){
	// We could assume that every empty cell has at least 2 options, therefore no need for filtering (2+)-option cells
	int A;
	int cellA;
	int B;
	int cellB;
	int C;
	int cellC;
	// Find twins per column
	for (int c = 0; c < boardSize; c++)
	{
		// Compare every 2 distenct cells
		for (int i = 0; i < boardSize-2; i++)
		{
			A = i*boardSize + c;
			cellA = board[A];
			if (isEmpty(cellA)){ //avoid already set cells
				cellA = getCellOptions(cellA);
				for (int j = i+1; j < boardSize-1; j++)
				{
					B = j*boardSize + c;
					cellB = board[B];
					if (isEmpty(cellB)){ //avoid already set cells
						cellB = getCellOptions(cellB);
						for (int k = j+1; k < boardSize; k++)
						{
							C = k*boardSize + c;
							cellC = board[C];
							if (isEmpty(cellC)){ //avoid already set cells
								cellC = getCellOptions(cellC);
								// cellA, cellB and cellC are triplets if (cellA & cellB & cellC) has exactly 3 options
								int options = cellA & cellB & cellC;
								if (bitCount(options) >= 3){
									int optionsUnion = 0; // get all options for all left cells
									for (int t = 0; t < boardSize; t++)
									{
										if (t != i && t != j){
											int tempCell = board[t*boardSize + c];
											optionsUnion = optionsUnion | getCellOptions(tempCell);
										}
									}
									options = options & ~optionsUnion; // remove options that are in the union
									if (bitCount(options) == 3){
										if (options != cellA){
											setOptions(board, A, options);
											choicesChanged = true;
										}
										if (options != cellB){
											setOptions(board, B, options);
											choicesChanged = true;
										}
										if (options != cellC){
											setOptions(board, C, options);
											choicesChanged = true;
										}
									}
								}
							}
						}
					}
				}
			}
		}
	}
}

void twins(int *board, int boardSize, bool &choicesChanged) {
	twinsInRow(board, boardSize, choicesChanged);
	twinsInBox(board, boardSize, choicesChanged);//Some overlapping work will occur
	twinsInColumn(board, boardSize, choicesChanged);
}

void triplets(int *board, int boardSize, bool &choicesChanged) {
	tripletsInRow(board, boardSize, choicesChanged);
	tripletsInBox(board, boardSize, choicesChanged);//Some overlapping work will occur
	tripletsInColumn(board, boardSize, choicesChanged);
}

bool humanistic(int *board, int boardSize, int n) {
	/* return false if board has no solution
		 otherwise (i.e. if solution found or algorithm makes no more changes) return true */

	//in some step of algorithm, a cell was written to with its final value
	bool cellChanged = true;
	//in some step of algorithm, choices were eliminated from some cell
	bool choicesChanged = true;
	while (cellChanged || choicesChanged) {
		if (cellChanged) {
			cellChanged = false;
			if (!elimination(board, boardSize, cellChanged, n)) return false;
			if (cellChanged) continue;
		}
		if (choicesChanged) {
			loneRanger(board, boardSize, cellChanged, n);
			if (cellChanged) continue;
			choicesChanged = false;
			boxElimination(board, boardSize, choicesChanged, n);
			if (choicesChanged) continue;
			twins(board, boardSize, choicesChanged);
			if (choicesChanged) continue;
			triplets(board, boardSize, choicesChanged);
		}
	}
	return true;
}


int *bruteForce(int *board, int boardSize, int n) {
  int totalSquares = boardSize * boardSize;
  for (int i=0; i < totalSquares; i++) {
    int value = board[i];
    if (!(value % (1<<VALUEBITS))) { //cell is empty
      value = value >> VALUEBITS;
      int choice = 0;
      //printBoard(board, boardSize);
      //printf("row: %d, col: %d\n", i/boardSize, i%boardSize);
      while (value) {
        value = value>>1;
        choice++;
        if (value % 2) {
          //printf("choice: %d\n", choice);
          int *newBoard = (int *)calloc(totalSquares, sizeof(int));
          memcpy(newBoard, board, totalSquares * sizeof(int));
          newBoard[i] = (1 << (VALUEBITS + choice)) + choice;
          //printBoard(newBoard, boardSize);
          eliminateChoices(newBoard, boardSize, i / boardSize, i % boardSize, n);
          if (!humanistic(board, boardSize, n)){
            //no solution exists
            return NULL;
          } 
          int *solution = bruteForce(newBoard, boardSize, n);
          if (solution) return solution; //if a solution exists, return it
          free(newBoard);
        }
      }
      return NULL; //there is no solution for the given board
    }
  }
  return board;
}

double cudaSudoku(int *board, int boardSize, int n) {
	//do initial choice elimination based on given board
	initialChoiceElm(board, boardSize, n);

	double startTime = CycleTimer::currentSeconds();

	if (!humanistic(board, boardSize, n)) {
		//no solution exists
		board = NULL;
	} else board = bruteForce(board, boardSize, n);

	double endTime = CycleTimer::currentSeconds();

	return endTime - startTime;
}



void printCudaInfo()
{
		// for fun, just print out some stats on the machine

		int deviceCount = 0;
		hipError_t err = hipGetDeviceCount(&deviceCount);

		printf("---------------------------------------------------------\n");
		printf("Found %d CUDA devices\n", deviceCount);

		for (int i=0; i<deviceCount; i++)
		{
				hipDeviceProp_t deviceProps;
				hipGetDeviceProperties(&deviceProps, i);
				printf("Device %d: %s\n", i, deviceProps.name);
				printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
				printf("   Global mem: %.0f MB\n",
							 static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
				printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
		}
		printf("---------------------------------------------------------\n"); 
}
